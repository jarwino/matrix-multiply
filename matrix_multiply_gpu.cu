#include "hip/hip_runtime.h"
#include "matrix_multiply_gpu.h"

__global__ void multiply_matrices(double* a_gpu, double* b_gpu, double* c_gpu) {

  int i = blockDim.y * blockIdx.y + threadIdx.y;
  int j = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < N && j < P) {
      double sum = 0;
      for (int k = 0; k < M; k++) {
        sum += a_gpu[i * M + k] * b_gpu[k * P + j];
      }
      c_gpu[i * P + j] = sum;
  }
}

void multiply_matrices_gpu(double a[N * M], double b[M * P], double c[N * P]) {

  double* a_gpu;
  double* b_gpu;
  double* c_gpu;

  hipMalloc(&a_gpu, N * M * sizeof(double));
  hipMalloc(&b_gpu, M * P * sizeof(double));
  hipMalloc(&c_gpu, N * P * sizeof(double));

  hipMemcpy(a_gpu, a, N * M * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(b_gpu, b, M * P * sizeof(double), hipMemcpyHostToDevice);

  int threadsPerBlock1D = 16;

  dim3 numBlocks((P + threadsPerBlock1D - 1) / threadsPerBlock1D, (N + threadsPerBlock1D - 1) / threadsPerBlock1D, 1);
  dim3 threadsPerBlock(threadsPerBlock1D, threadsPerBlock1D, 1);

  multiply_matrices<<<numBlocks, threadsPerBlock>>>(a_gpu, b_gpu, c_gpu);

  hipMemcpy(c, c_gpu, N * P * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(a_gpu);
  hipFree(b_gpu);
  hipFree(c_gpu);

}
